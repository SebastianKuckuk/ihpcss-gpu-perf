#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>

#include "../util.h"
#include "../cuda-util.h"
#include "strided-stream-util.h"


__global__ void stream(size_t nx, const tpe *__restrict__ src, tpe *__restrict__ dest, unsigned int stride) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < nx; i += gridDim.x * blockDim.x)
        dest[stride * i] = src[stride * i] + 1;
}


int main(int argc, char *argv[]) {
    size_t nx, nItWarmUp, nIt;
    unsigned int stride;
    parseCLA_1d(argc, argv, nx, nItWarmUp, nIt, stride);

    tpe *src, *dest;
    checkCudaError(hipHostMalloc((void **) &src, sizeof(tpe) * stride * nx));
    checkCudaError(hipHostMalloc((void **) &dest, sizeof(tpe) * stride * nx));

    // init
    initStream(src, nx, stride);

    tpe *d_src, *d_dest;
    checkCudaError(hipMalloc((void **) &d_src, sizeof(tpe) * stride * nx));
    checkCudaError(hipMalloc((void **) &d_dest, sizeof(tpe) * stride * nx));

    checkCudaError(hipMemcpy(d_src, src, sizeof(tpe) * stride * nx, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_dest, dest, sizeof(tpe) * stride * nx, hipMemcpyHostToDevice));

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t numSM = deviceProp.multiProcessorCount;

    dim3 blockSize(256);
    dim3 numBlocks(std::min(32 * numSM, ceilingDivide(nx, blockSize.x)));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        stream<<<numBlocks, blockSize>>>(nx, d_src, d_dest, stride);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        stream<<<numBlocks, blockSize>>>(nx, d_src, d_dest, stride);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    printStats(end - start, nx, nIt, 2 * sizeof(tpe), 1);

    checkCudaError(hipMemcpy(src, d_src, sizeof(tpe) * stride * nx, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(dest, d_dest, sizeof(tpe) * stride * nx, hipMemcpyDeviceToHost));

    // check solution
    checkSolutionStream(src, nx, nIt + nItWarmUp, stride);

    checkCudaError(hipFree(d_src));
    checkCudaError(hipFree(d_dest));

    checkCudaError(hipHostFree(src));
    checkCudaError(hipHostFree(dest));

    return 0;
}
