#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>

#include "../util.h"
#include "../cuda-util.h"
#include "strided-fma-util.h"


__global__ void fma(size_t nx, const tpe *__restrict__ src, tpe *__restrict__ dest, unsigned int stride) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < stride * nx; i += gridDim.x * blockDim.x) {
        if (0 == i % stride) {
            auto acc = src[i / stride];

            for (auto j = 0; j < numRepetitions; ++j)
                acc = (tpe)0.5 * acc + (tpe)1;

            dest[i / stride] = acc;
        }
    }
}


int main(int argc, char *argv[]) {
    size_t nx, nItWarmUp, nIt;
    unsigned int stride;
    parseCLA_1d(argc, argv, nx, nItWarmUp, nIt, stride);

    tpe *src, *dest;
    checkCudaError(hipHostMalloc((void **) &src, sizeof(tpe) * nx));
    checkCudaError(hipHostMalloc((void **) &dest, sizeof(tpe) * nx));

    // init
    initFMA(src, nx);

    tpe *d_src, *d_dest;
    checkCudaError(hipMalloc((void **) &d_src, sizeof(tpe) * nx));
    checkCudaError(hipMalloc((void **) &d_dest, sizeof(tpe) * nx));

    checkCudaError(hipMemcpy(d_src, src, sizeof(tpe) * nx, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_dest, dest, sizeof(tpe) * nx, hipMemcpyHostToDevice));

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t numSM = deviceProp.multiProcessorCount;

    dim3 blockSize(256);
    dim3 numBlocks(std::min(32 * numSM, ceilingDivide(nx, blockSize.x)));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        fma<<<numBlocks, blockSize>>>(nx, d_src, d_dest, stride);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        fma<<<numBlocks, blockSize>>>(nx, d_src, d_dest, stride);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    printStats(end - start, nx, nIt, 2 * sizeof(tpe), 2 * numRepetitions);

    checkCudaError(hipMemcpy(src, d_src, sizeof(tpe) * nx, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(dest, d_dest, sizeof(tpe) * nx, hipMemcpyDeviceToHost));

    // check solution
    checkSolutionFMA(src, nx, nIt + nItWarmUp);

    checkCudaError(hipFree(d_src));
    checkCudaError(hipFree(d_dest));

    checkCudaError(hipHostFree(src));
    checkCudaError(hipHostFree(dest));

    return 0;
}
