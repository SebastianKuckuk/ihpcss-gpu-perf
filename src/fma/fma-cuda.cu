#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>

#include "../util.h"
#include "../cuda-util.h"
#include "fma-util.h"


__global__ void fma(size_t nx, const tpe *__restrict__ src, tpe *__restrict__ dest) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < nx; i += gridDim.x * blockDim.x) {
        auto acc = src[i];
        for (auto j = 0; j < numRepetitions; ++j)
            acc = (tpe)0.5 * acc + (tpe)1;

        dest[i] = acc;
    }
}


int main(int argc, char *argv[]) {
    size_t nx, nItWarmUp, nIt;
    parseCLA_1d(argc, argv, nx, nItWarmUp, nIt);

    tpe *src, *dest;
    checkCudaError(hipHostMalloc((void **) &src, sizeof(tpe) * nx));
    checkCudaError(hipHostMalloc((void **) &dest, sizeof(tpe) * nx));

    // init
    initFMA(src, nx);

    tpe *d_src, *d_dest;
    checkCudaError(hipMalloc((void **) &d_src, sizeof(tpe) * nx));
    checkCudaError(hipMalloc((void **) &d_dest, sizeof(tpe) * nx));

    checkCudaError(hipMemcpy(d_src, src, sizeof(tpe) * nx, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_dest, dest, sizeof(tpe) * nx, hipMemcpyHostToDevice));

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t numSM = deviceProp.multiProcessorCount;

    dim3 blockSize(256);
    dim3 numBlocks(std::min(32 * numSM, ceilingDivide(nx, blockSize.x)));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        fma<<<numBlocks, blockSize>>>(nx, d_src, d_dest);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        fma<<<numBlocks, blockSize>>>(nx, d_src, d_dest);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    printStats(end - start, nx, nIt, 2 * sizeof(tpe), 2 * numRepetitions);

    checkCudaError(hipMemcpy(src, d_src, sizeof(tpe) * nx, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(dest, d_dest, sizeof(tpe) * nx, hipMemcpyDeviceToHost));

    // check solution
    checkSolutionFMA(src, nx, nIt + nItWarmUp);

    checkCudaError(hipFree(d_src));
    checkCudaError(hipFree(d_dest));

    checkCudaError(hipHostFree(src));
    checkCudaError(hipHostFree(dest));

    return 0;
}
