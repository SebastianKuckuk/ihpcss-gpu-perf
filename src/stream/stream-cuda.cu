#include "hip/hip_runtime.h"
#include <chrono>
#include <iostream>

#include "../util.h"
#include "../cuda-util.h"
#include "stream-util.h"


__global__ void stream(size_t nx, const tpe *__restrict__ src, tpe *__restrict__ dest) {
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < nx; i += gridDim.x * blockDim.x)
        dest[i] = src[i] + 1;
}


int main(int argc, char *argv[]) {
    size_t nx, nItWarmUp, nIt;
    parseCLA_1d(argc, argv, nx, nItWarmUp, nIt);

    tpe *src, *dest;
    checkCudaError(hipHostMalloc((void **) &src, sizeof(tpe) * nx));
    checkCudaError(hipHostMalloc((void **) &dest, sizeof(tpe) * nx));

    // init
    initStream(src, nx);

    tpe *d_src, *d_dest;
    checkCudaError(hipMalloc((void **) &d_src, sizeof(tpe) * nx));
    checkCudaError(hipMalloc((void **) &d_dest, sizeof(tpe) * nx));

    checkCudaError(hipMemcpy(d_src, src, sizeof(tpe) * nx, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_dest, dest, sizeof(tpe) * nx, hipMemcpyHostToDevice));

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    size_t numSM = deviceProp.multiProcessorCount;

    dim3 blockSize(256);
    dim3 numBlocks(std::min(32 * numSM, ceilingDivide(nx, blockSize.x)));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        stream<<<numBlocks, blockSize>>>(nx, d_src, d_dest);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        stream<<<numBlocks, blockSize>>>(nx, d_src, d_dest);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    printStats(end - start, nx, nIt, 2 * sizeof(tpe), 1);

    checkCudaError(hipMemcpy(src, d_src, sizeof(tpe) * nx, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(dest, d_dest, sizeof(tpe) * nx, hipMemcpyDeviceToHost));

    // check solution
    checkSolutionStream(src, nx, nIt + nItWarmUp);

    checkCudaError(hipFree(d_src));
    checkCudaError(hipFree(d_dest));

    checkCudaError(hipHostFree(src));
    checkCudaError(hipHostFree(dest));

    return 0;
}
